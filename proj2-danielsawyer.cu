#include "hip/hip_runtime.h"
/* ==================================================================
	Programmer: Daniel Sawyer (danielsawyer@mail.usf.edu)
	The basic SDH algorithm implementation for 3D data
	To compile: nvcc proj2-danielsawyer.cu -o proj2 in the rc machines
   ==================================================================
*/

#include <stdio.h>
#include <stdlib.h>
#include <math.h>
#include <sys/time.h>

//MY INCLUDES
#include <iostream>
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>


#define BOX_SIZE	23000 /* size of the data box on one dimension            */

/* descriptors for single atom in the tree */
typedef struct atomdesc {
	double x_pos;
	double y_pos;
	double z_pos;
} atom;

typedef struct hist_entry{
	//float min;
	//float max;
	long long d_cnt;   /* need a long long type as the count might be huge */
} bucket;


bucket * histogram;		/* list of all buckets in the histogram   */
long long	PDH_acnt;	/* total number of data points            */
int num_buckets;		/* total number of buckets in the histogram */
double   PDH_res;		/* value of w                             */
atom * atom_list;		/* list of all data points                */
int blockSize;

/* These are for an old way of tracking time */
struct timezone Idunno;
struct timeval startTime, endTime;


/* 
	distance of two points in the atom_list 
*/
double p2p_distance(int ind1, int ind2) {
	
	double x1 = atom_list[ind1].x_pos;
	double x2 = atom_list[ind2].x_pos;
	double y1 = atom_list[ind1].y_pos;
	double y2 = atom_list[ind2].y_pos;
	double z1 = atom_list[ind1].z_pos;
	double z2 = atom_list[ind2].z_pos;
		
	return sqrt((x1 - x2)*(x1-x2) + (y1 - y2)*(y1 - y2) + (z1 - z2)*(z1 - z2));
}


/* 
	set a checkpoint and show the (natural) running time in seconds 
*/
double report_running_time() {
	long sec_diff, usec_diff;
	gettimeofday(&endTime, &Idunno);
	sec_diff = endTime.tv_sec - startTime.tv_sec;
	usec_diff= endTime.tv_usec-startTime.tv_usec;
	if(usec_diff < 0) {
		sec_diff --;
		usec_diff += 1000000;
	}
	printf("Running time for CPU version: %ld.%06ld\n", sec_diff, usec_diff);
	return (double)(sec_diff*1.0 + usec_diff/1000000.0);
}
//overloaded to show GPU time
double report_running_time(int blah) {
	long sec_diff, usec_diff;
	gettimeofday(&endTime, &Idunno);
	sec_diff = endTime.tv_sec - startTime.tv_sec;
	usec_diff= endTime.tv_usec-startTime.tv_usec;
	if(usec_diff < 0) {
		sec_diff --;
		usec_diff += 1000000;
	}
	printf("\nRunning time for GPU version: %ld.%06ld\n", sec_diff, usec_diff);
	return (double)(sec_diff*1.0 + usec_diff/1000000.0);
}

/* 
	print the counts in all buckets of the histogram 
*/
void output_histogram(){
	int i; 
	long long total_cnt = 0;
	for(i=0; i< num_buckets; i++) {
		if(i%5 == 0) /* we print 5 buckets in a row */
			printf("\n%02d: ", i);
		printf("%15lld ", histogram[i].d_cnt);
		total_cnt += histogram[i].d_cnt;
	  	/* we also want to make sure the total distance count is correct */
		if(i == num_buckets - 1)	
			printf("\n T:%lld \n", total_cnt);
		else printf("| ");
	}
}
//overloaded taking 1 arg
void output_histogram(bucket* histogram1){
	int i; 
	long long total_cnt = 0;
	for(i=0; i< num_buckets; i++) {
		if(i%5 == 0) /* we print 5 buckets in a row */
			printf("\n%02d: ", i);
		printf("%15lld ", histogram1[i].d_cnt);
		total_cnt += histogram1[i].d_cnt;
	  	/* we also want to make sure the total distance count is correct */
		if(i == num_buckets - 1)	
			printf("\n T:%lld \n", total_cnt);
		else printf("| ");
	}
}
//overloaded taking 2 args
void output_histogram(bucket* histogram1, bucket* histogram2){
	int i; 
	long long total_cnt = 0, total_cnt2 = 0;
	for(i=0; i< num_buckets; i++) {
		if(i%5 == 0) /* we print 5 buckets in a row */
			printf("\n%02d: ", i);
		printf("%15lld ", abs(histogram1[i].d_cnt - histogram2[i].d_cnt));
		total_cnt += histogram1[i].d_cnt;
		total_cnt2 += histogram2[i].d_cnt;
	  	/* we also want to make sure the total distance count is correct */
		if(i == num_buckets - 1)	
			printf("\n T:%llu \n", abs(total_cnt - total_cnt2));
		else printf("| ");
	}
}

/* 
	brute-force SDH solution in a single CPU thread 
*/
int PDH_baseline() {
	int i, j, h_pos;
	double dist;
	
	for(i = 0; i < PDH_acnt; i++) {
		for(j = i+1; j < PDH_acnt; j++) {
			dist = p2p_distance(i,j);
			h_pos = (int) (dist / PDH_res);
			histogram[h_pos].d_cnt++;
		} 
	}
	return 0;
}

__device__ double pdist(atom &L, atom &R) {

	double x = L.x_pos - R.x_pos;
	double y = L.y_pos - R.y_pos;
	double z = L.z_pos - R.z_pos;

	return sqrt(x*x + y*y + z*z);
}


/**********	KERNEL STUFF **********/

//NAIVE
__global__ void Naive(atom* dev_points, bucket *dev_Hist, int d_PDH_acnt, int d_PDH_res) {

	float dist = 0.0;
	int h_ind = 0;
	atom currentPt;
	unsigned long long int tid = threadIdx.x + blockIdx.x * blockDim.x;

	if(tid < d_PDH_acnt) {
		
		currentPt = dev_points[tid];

		for(int i = tid+1; i < d_PDH_acnt; i++) {

			dist = pdist(currentPt, dev_points[i]);

			h_ind = (int)dist / d_PDH_res;

			atomicAdd((unsigned long long int*)&dev_Hist[h_ind].d_cnt, 1);
		}
	}
}

//CUDA KERNEL ALGO 3
__global__ void PDH_Algo3(atom *d_atom_list, bucket *d_histogram, long long d_PDH_acnt, double d_PDH_res, int nbuckets, int nblocks) {

	//vars
	register double dist;
	register int i, j, h_pos, tx = threadIdx.x, dx = blockDim.x, bx = blockIdx.x;
	register long long tid = threadIdx.x + blockDim.x * blockIdx.x;
	register atom L;

	//shared vars
	extern __shared__ atom smem[];
	atom* R = (atom*)smem;
	bucket* s_histogram = (bucket*)&smem[dx];
	
	//initialize histogram to 0 and load L in parallel
	for(i = tx; i < nbuckets; i += dx)
		s_histogram[i].d_cnt = 0;

	L = d_atom_list[tid];
	__syncthreads();

	//runs through other blocks
	for(i = bx + 1; i < nblocks; i++) {

		//loads tile of atoms into shared mem
		R[tx] = d_atom_list[tx + i*dx];
		__syncthreads();

		//runs through the shared mem, saves to hist
		if(i*dx < d_PDH_acnt)
		for(j = 0; j < dx; j++) {
			
			if(j + i*dx < d_PDH_acnt) {

				dist = pdist(L, R[j]);

				h_pos = (int)(dist/d_PDH_res);

				atomicAdd((unsigned long long int*)&s_histogram[h_pos].d_cnt,1);
			}
		}
		__syncthreads();
	}

	//loads currents block into shared
	R[tx] = L;
	__syncthreads();

	//calcs distances for current block
	if(tid < d_PDH_acnt)
	for(j = tx + 1; j < dx; j++) {

		if(j + bx*dx < d_PDH_acnt) {

			dist = pdist(L, R[j]);

			h_pos = (int)(dist/d_PDH_res);

			atomicAdd((unsigned long long int*)&s_histogram[h_pos].d_cnt,1);
		}
	}
	__syncthreads();

	//saves back to global
	for(i = tx; i < nbuckets; i += dx)
		atomicAdd((unsigned long long int*)&d_histogram[i].d_cnt,s_histogram[i].d_cnt);
}

/**********	KERNEL STUFF **********/

//preps for kernel call
float CudaPrep(bucket * histogram2) {

	//sizes of atom and bucket arrays
	int size_atom = sizeof(atom)*PDH_acnt;
	int size_hist = sizeof(bucket)*num_buckets;

	//grid and block sizes
	int dev = 0;
	hipSetDevice(dev);
    hipDeviceProp_t deviceProp;
    hipGetDeviceProperties(&deviceProp, dev);
	
	dim3 threads(blockSize);
	//dim3 threads(deviceProp.warpSize);
	dim3 grid(ceil((float)PDH_acnt/threads.x));

	//Device Vars
	bucket *d_histogram;
	atom *d_atom_list;
	int num_blocks = ceil((float)PDH_acnt/threads.x);

	//Allocate device memory
	hipMalloc((void **) &d_histogram, size_hist);
	hipMalloc((void**) &d_atom_list, size_atom);

	//Copy to device
	hipMemcpy(d_atom_list, atom_list, size_atom, hipMemcpyHostToDevice);
	hipMemset(d_histogram, 0, size_hist);

	//kernel execution time crap
	float elapsedTime;
	hipEvent_t start, stop;
	hipEventCreate(&start);
	hipEventCreate(&stop);
	hipEventRecord(start, 0);
	
	//run cuda kernel
	//Naive<<<grid, threads>>>(d_atom_list, d_histogram, PDH_acnt, PDH_res);
	PDH_Algo3<<<grid, threads, num_buckets*sizeof(bucket) + threads.x*sizeof(atom)>>>(d_atom_list, d_histogram, PDH_acnt, PDH_res, num_buckets, num_blocks);

	//kernel execution stop
	//hipDeviceSynchronize();
	hipEventRecord(stop, 0);
	hipEventSynchronize(stop);
	hipEventElapsedTime(&elapsedTime, start, stop);
	hipEventDestroy(start);
	hipEventDestroy(stop);

	//copy new gpu histogram back to host from device
	hipMemcpy(histogram2, d_histogram, size_hist, hipMemcpyDeviceToHost);

	//free device memory
	hipFree(d_histogram); hipFree(d_atom_list);

	return elapsedTime;
}

int main(int argc, char **argv)
{
	if(argc < 4) {

		printf("\nError, too few arguments. Usage: ./proj2 {#of_samples} {bucket_width} {block_size}.\n");
		return -1;
	}

	int i;

	PDH_acnt = atoi(argv[1]);
	PDH_res	 = atof(argv[2]);
	blockSize = atof(argv[3]);

	num_buckets = (int)(BOX_SIZE * 1.732 / PDH_res) + 1;
	histogram = (bucket *)malloc(sizeof(bucket)*num_buckets);

	atom_list = (atom *)malloc(sizeof(atom)*PDH_acnt);

	
	srand(1);
	/* generate data following a uniform distribution */
	for(i = 0;  i < PDH_acnt; i++) {
		atom_list[i].x_pos = ((double)(rand()) / RAND_MAX) * BOX_SIZE;
		atom_list[i].y_pos = ((double)(rand()) / RAND_MAX) * BOX_SIZE;
		atom_list[i].z_pos = ((double)(rand()) / RAND_MAX) * BOX_SIZE;
	}
	/* start counting time */
	gettimeofday(&startTime, &Idunno);
	
	/* call CPU single thread version to compute the histogram */
	PDH_baseline();
	
	/* check the total running time */ 
	report_running_time();
	
	/* print out the histogram */
	output_histogram();


	/* NEW SHIT */

	//New histogram that will come from the device
	bucket *histogram2 = (bucket*)malloc(sizeof(bucket)*num_buckets);
	memset(histogram2, 0, size_hist);

	//start time
	gettimeofday(&startTime, &Idunno);

	//run on GPU
	float elapsedTime = CudaPrep(histogram2);

	//check runtime
	report_running_time(1);

	//print device histogram
	output_histogram(histogram2);

	//Difference between cpu and gpu
	printf("\nCPU vs GPU Histogram Differences\n");
	output_histogram(histogram, histogram2);

	//Free memory.
	free(histogram); free(atom_list);

	printf("\n******** Total Running Time of Kernel = %0.5f ms *******\n", elapsedTime);
	printf("\n******** Total Running Time of Kernel = %0.5f sec *******\n", elapsedTime/1000);
	
	return 0;
}
