/* ==================================================================
	Programmer: Daniel Sawyer (danielsawyer@mail.usf.edu)
	The basic SDH algorithm implementation for 3D data
	To compile: nvcc proj1-danielsawyer.cu -o SDH in the rc machines
   ==================================================================
*/

#include <stdio.h>
#include <stdlib.h>
#include <math.h>
#include <sys/time.h>

//MY INCLUDES
#include <iostream>
#include <hip/hip_runtime.h>



#define BOX_SIZE	23000 /* size of the data box on one dimension            */

/* descriptors for single atom in the tree */
typedef struct atomdesc {
	double x_pos;
	double y_pos;
	double z_pos;
} atom;

typedef struct hist_entry{
	//float min;
	//float max;
	long long d_cnt;   /* need a long long type as the count might be huge */
} bucket;


bucket * histogram;		/* list of all buckets in the histogram   */
long long	PDH_acnt;	/* total number of data points            */
int num_buckets;		/* total number of buckets in the histogram */
double   PDH_res;		/* value of w                             */
atom * atom_list;		/* list of all data points                */

/* These are for an old way of tracking time */
struct timezone Idunno;	
struct timeval startTime, endTime;


/* 
	distance of two points in the atom_list 
*/
double p2p_distance(int ind1, int ind2) {
	
	double x1 = atom_list[ind1].x_pos;
	double x2 = atom_list[ind2].x_pos;
	double y1 = atom_list[ind1].y_pos;
	double y2 = atom_list[ind2].y_pos;
	double z1 = atom_list[ind1].z_pos;
	double z2 = atom_list[ind2].z_pos;
		
	return sqrt((x1 - x2)*(x1-x2) + (y1 - y2)*(y1 - y2) + (z1 - z2)*(z1 - z2));
}


/* 
	set a checkpoint and show the (natural) running time in seconds 
*/
double report_running_time() {
	long sec_diff, usec_diff;
	gettimeofday(&endTime, &Idunno);
	sec_diff = endTime.tv_sec - startTime.tv_sec;
	usec_diff= endTime.tv_usec-startTime.tv_usec;
	if(usec_diff < 0) {
		sec_diff --;
		usec_diff += 1000000;
	}
	printf("Running time for CPU version: %ld.%06ld\n", sec_diff, usec_diff);
	return (double)(sec_diff*1.0 + usec_diff/1000000.0);
}
//overloaded to show GPU time
double report_running_time(int blah) {
	long sec_diff, usec_diff;
	gettimeofday(&endTime, &Idunno);
	sec_diff = endTime.tv_sec - startTime.tv_sec;
	usec_diff= endTime.tv_usec-startTime.tv_usec;
	if(usec_diff < 0) {
		sec_diff --;
		usec_diff += 1000000;
	}
	printf("\nRunning time for GPU version: %ld.%06ld\n", sec_diff, usec_diff);
	return (double)(sec_diff*1.0 + usec_diff/1000000.0);
}

/* 
	print the counts in all buckets of the histogram 
*/
void output_histogram(){
	int i; 
	long long total_cnt = 0;
	for(i=0; i< num_buckets; i++) {
		if(i%5 == 0) /* we print 5 buckets in a row */
			printf("\n%02d: ", i);
		printf("%15lld ", histogram[i].d_cnt);
		total_cnt += histogram[i].d_cnt;
	  	/* we also want to make sure the total distance count is correct */
		if(i == num_buckets - 1)	
			printf("\n T:%lld \n", total_cnt);
		else printf("| ");
	}
}
//overloaded taking 1 arg
void output_histogram(bucket* histogram1){
	int i; 
	long long total_cnt = 0;
	for(i=0; i< num_buckets; i++) {
		if(i%5 == 0) /* we print 5 buckets in a row */
			printf("\n%02d: ", i);
		printf("%15lld ", histogram1[i].d_cnt);
		total_cnt += histogram1[i].d_cnt;
	  	/* we also want to make sure the total distance count is correct */
		if(i == num_buckets - 1)	
			printf("\n T:%lld \n", total_cnt);
		else printf("| ");
	}
}
//overloaded taking 2 args
void output_histogram(bucket* histogram1, bucket* histogram2){
	int i; 
	long long total_cnt = 0, total_cnt2 = 0;
	for(i=0; i< num_buckets; i++) {
		if(i%5 == 0) /* we print 5 buckets in a row */
			printf("\n%02d: ", i);
		printf("%15lld ", abs(histogram1[i].d_cnt - histogram2[i].d_cnt));
		total_cnt += histogram1[i].d_cnt;
		total_cnt2 += histogram2[i].d_cnt;
	  	/* we also want to make sure the total distance count is correct */
		if(i == num_buckets - 1)	
			printf("\n T:%lld \n", abs(total_cnt - total_cnt2));
		else printf("| ");
	}
}

/* 
	brute-force SDH solution in a single CPU thread 
*/
int PDH_baseline() {
	int i, j, h_pos;
	double dist;
	
	for(i = 0; i < PDH_acnt; i++) {
		for(j = i+1; j < PDH_acnt; j++) {
			dist = p2p_distance(i,j);
			h_pos = (int) (dist / PDH_res);
			histogram[h_pos].d_cnt++;
		} 
	}
	return 0;
}

//CUDA KERNEL FOR SDH
__global__ void PDH_Cuda(atom *d_atom_list, bucket *d_histogram, long long d_PDH_acnt, double d_PDH_res) {

	double dist;
	int i, j, h_pos;

	i = threadIdx.x + blockDim.x * blockIdx.x;
	//if(i == 0) printf("\nTHE I VALUE = 0\n");
	//if(i == 0) printf("\nwarpSize = %d\n", warpSize);
	for(j = i+1; j < d_PDH_acnt; ++j) {

		dist = sqrt( (d_atom_list[i].x_pos - d_atom_list[j].x_pos)*(d_atom_list[i].x_pos - d_atom_list[j].x_pos) +
					 (d_atom_list[i].y_pos - d_atom_list[j].y_pos)*(d_atom_list[i].y_pos - d_atom_list[j].y_pos) +
					 (d_atom_list[i].z_pos - d_atom_list[j].z_pos)*(d_atom_list[i].z_pos - d_atom_list[j].z_pos) );
		h_pos = (int)(dist/d_PDH_res);		
		atomicAdd((unsigned long long int*)&d_histogram[h_pos].d_cnt,1);
		//d_histogram[h_pos].d_cnt++;
		//__syncthreads();
	}
}

void CudaPrep(bucket * histogram2) {

	//sizes of atom and bucket arrays
	int size_atom = sizeof(atom)*PDH_acnt;
	int size_hist = sizeof(bucket)*num_buckets;

	//grid and block sizes
	int dev = 0;
	hipSetDevice(dev);
    hipDeviceProp_t deviceProp;
    hipGetDeviceProperties(&deviceProp, dev);
	//printf("\nWARP = %d\n", deviceProp.warpSize);
	dim3 threads(deviceProp.warpSize);
	dim3 grid(ceil((float)PDH_acnt/threads.x));

	//Device Vars
	bucket *d_histogram;
	atom *d_atom_list;

	//Allocate device memory
	hipMalloc((void **) &d_histogram, size_hist);
	hipMalloc((void**) &d_atom_list, size_atom);

	//Copy to device
	hipMemcpy(d_atom_list, atom_list, size_atom, hipMemcpyHostToDevice);
	hipMemset(d_histogram, 0, size_hist);

	//run cuda kernel
	PDH_Cuda<<<grid,threads>>>(d_atom_list, d_histogram, PDH_acnt, PDH_res);

	//copy new gpu histogram back to host from device
	hipMemcpy(histogram2, d_histogram, size_hist, hipMemcpyDeviceToHost);

	//free device memory
	hipFree(d_histogram); hipFree(d_atom_list);
}

int main(int argc, char **argv)
{
	int i;

	PDH_acnt = atoi(argv[1]);
	PDH_res	 = atof(argv[2]);
	//printf("args are %d and %f\n", PDH_acnt, PDH_res);

	num_buckets = (int)(BOX_SIZE * 1.732 / PDH_res) + 1;
	histogram = (bucket *)malloc(sizeof(bucket)*num_buckets);

	atom_list = (atom *)malloc(sizeof(atom)*PDH_acnt);

	
	srand(1);
	/* generate data following a uniform distribution */
	for(i = 0;  i < PDH_acnt; i++) {
		atom_list[i].x_pos = ((double)(rand()) / RAND_MAX) * BOX_SIZE;
		atom_list[i].y_pos = ((double)(rand()) / RAND_MAX) * BOX_SIZE;
		atom_list[i].z_pos = ((double)(rand()) / RAND_MAX) * BOX_SIZE;
	}
	/* start counting time */
	gettimeofday(&startTime, &Idunno);
	
	/* call CPU single thread version to compute the histogram */
	PDH_baseline();
	
	/* check the total running time */ 
	report_running_time();
	
	/* print out the histogram */
	output_histogram();


	/* NEW SHIT */

	//New histogram that will come from the device
	bucket *histogram2 = (bucket*)malloc(sizeof(bucket)*num_buckets);
	//memset(histogram2, 0, size_hist);

	//start time
	gettimeofday(&startTime, &Idunno);

	//run on GPU
	CudaPrep(histogram2);

	//check runtime
	report_running_time(1);

	//print device histogram
	output_histogram(histogram2);

	//Difference between cpu and gpu
	printf("\nCPU vs GPU Histogram Differences\n");
	output_histogram(histogram, histogram2);

	//Free memory.
	free(histogram); free(atom_list);
	
	return 0;
}
